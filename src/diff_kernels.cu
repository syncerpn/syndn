#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "diff_layer.h"
#include "blas.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void diff_fill_skipped_channel_with_zero_kernel(int batch, int w, int h, int c, int* skip_list, int n, float* output, float* delta) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= batch * n * w * h) return;
    int iw = index % w;
    index = (index - iw) / w;
    int ih = index % h;
    index = (index - ih) / h;
    int ic = index % n;
    int ib = (index - ic) / n;

    int output_index = ib * w * h * c + skip_list[ic] * w * h + ih * w + iw;
    output[output_index] = 0;
    delta[output_index] = 0;
}

void diff_fill_skipped_channel_with_zero_gpu(int batch, int w, int h, int c, int* skip_list, int n, float* output, float* delta) {
    size_t N = batch * w * h * n;
    diff_fill_skipped_channel_with_zero_kernel<<<cuda_gridsize(N), BLOCK>>>(batch, w, h, c, skip_list, n, output, delta);
    check_error(hipPeekAtLastError());
}

__global__ void diff_fill_mask_layer_softmax_with_zero_kernel(int batch, int w, int h, int c, float* delta, float* mask_score, int classes, float* truth) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= batch * c * w * h) return;
    int iw = index % w;
    index = (index - iw) / w;
    int ih = index % h;
    index = (index - ih) / h;
    int ic = index % c;
    int ib = (index - ic) / c;

    int i;
    for (i = 0; i < classes; ++i) {
        if (truth[ib*classes + i] == 1) {
            break;
        }
    }
    // if (iw == 0 && ih == 0 && ic == 0) printf("%d %f\n", ib,  mask_score[i]);
    delta[index] *= mask_score[i];
}

void diff_fill_mask_layer_softmax_with_zero_gpu(int batch, int w, int h, int c, float* delta, float* mask_score, int classes, float* truth) {
    size_t N = batch * w * h * c;   
    diff_fill_mask_layer_softmax_with_zero_kernel<<<cuda_gridsize(N), BLOCK>>>(batch, w, h, c, delta, mask_score, classes, truth);
    check_error(hipPeekAtLastError());
}

void forward_diff_layer_gpu(const layer l, network net) {
	*(l.cost) = 0;
    float* truth = net.layers[l.input_layers[0]].output_gpu;
    float* learn = net.layers[l.input_layers[1]].output_gpu;

    switch (l.cost_type) {
        case SMOOTH:
            smooth_l1_gpu(l.batch * l.outputs, learn, truth, l.delta_gpu, l.output_gpu);
            break;
        case L1:
            l1_gpu(l.batch * l.outputs, learn, truth, l.delta_gpu, l.output_gpu);
            break;
        case WGAN:
            wgan_gpu(l.batch * l.outputs, learn, truth, l.delta_gpu, l.output_gpu);
            break;
        case SSE:
            l2_gpu(l.batch * l.outputs, learn, truth, l.delta_gpu, l.output_gpu);
            break;
        case SYMEXP:
            symexp_gpu(l.batch * l.outputs, learn, truth, l.delta_gpu, l.output_gpu);
            break;
        case LOGCOSH:
            logcosh_gpu(l.batch * l.outputs, learn, truth, l.delta_gpu, l.output_gpu);
            break;
        case MASKED:
        case SEG:
        default:
            fprintf(stderr, "Warning: unsupported cost type; use SSE instead\n");
            l2_gpu(l.batch * l.outputs, learn, truth, l.delta_gpu, l.output_gpu);
            break;
    }
    
    if (l.n > 0) {
        diff_fill_skipped_channel_with_zero_gpu(l.batch, l.w, l.h, l.c, l.indexes_gpu, l.n, l.output_gpu, l.delta_gpu);
    }

    if (l.mask_layer_softmax >= 0) {
        float* mask_score_gpu = net.layers[l.mask_layer_softmax].output_gpu;
        int classes = net.layers[l.mask_layer_softmax].inputs;
        diff_fill_mask_layer_softmax_with_zero_gpu(l.batch, l.w, l.h, l.c, l.delta_gpu, mask_score_gpu, classes, net.truth_gpu);
    }

    cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);

    int i;    
    for (i = 0; i < l.outputs * l.batch; ++i) {
        *(l.cost) += l.output[i];
    }

	*(l.cost) /= (l.outputs * l.batch - l.batch * l.n * l.w * l.h);
}

void backward_diff_layer_gpu(layer l, network net) {

    int index = l.input_layers[1];
    float *delta = net.layers[index].delta_gpu;
	axpy_gpu(l.inputs * l.batch, l.impact, l.delta_gpu, 1, delta, 1);
}