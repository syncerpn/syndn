#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
}

__global__ void make_shifting_weights_max_kernel(float *weights, int n, int size, float *tran_weights, float* q_coeff, int n_coeff, int zero_center)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float max = 0;
    for(i = 0; i < size; ++i){
        max = max < fabsf(weights[f*size + i]) ? fabsf(weights[f*size + i]) : max;
    }
    
    for(i = 0; i < size; ++i){
        if (weights[f*size + i] > 0) {
            float expo_factor = roundf(log2f(weights[f*size + i] / max));
            if (expo_factor > q_coeff[0]) expo_factor = q_coeff[0];
            else if (expo_factor < q_coeff[n_coeff-1]) {
                if (zero_center) {
                    tran_weights[f*size + i] = 0;
                    continue;
                } else {
                    expo_factor = q_coeff[n_coeff-1];
                }
            }
            tran_weights[f*size + i] = max * powf(2.f, expo_factor);
        } else {
            float expo_factor = roundf(log2f(-weights[f*size + i] / max));
            if (expo_factor > q_coeff[0]) expo_factor = q_coeff[0];
            else if (expo_factor < q_coeff[n_coeff-1]) {
                if (zero_center) {
                    tran_weights[f*size + i] = 0;
                    continue;
                } else {
                    expo_factor = q_coeff[n_coeff-1];
                }
            }
            tran_weights[f*size + i] = -max * powf(2.f, expo_factor);
        }
    }
}

void make_shifting_weights_max_gpu(float *weights, int n, int size, float *tran_weights, float* q_coeff, int n_coeff, int zero_center)
{
    make_shifting_weights_max_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, tran_weights, q_coeff, n_coeff, zero_center);
    check_error(hipPeekAtLastError());
}

__global__ void make_shifting_weights_mean_kernel(float *weights, int n, int size, float *tran_weights, float* q_coeff, int n_coeff, int zero_center)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += fabsf(weights[f*size + i]);
    }
    mean = mean / size;
    
    for(i = 0; i < size; ++i){
        if (weights[f*size + i] > 0) {
            float expo_factor = roundf(log2f(weights[f*size + i] / mean));
            if (expo_factor > q_coeff[0]) expo_factor = q_coeff[0];
            else if (expo_factor < q_coeff[n_coeff-1]) {
                if (zero_center) {
                    tran_weights[f*size + i] = 0;
                    continue;
                } else {
                    expo_factor = q_coeff[n_coeff-1];
                }
            }
            tran_weights[f*size + i] = mean * powf(2.f, expo_factor);
        } else {
            float expo_factor = roundf(log2f(-weights[f*size + i] / mean));
            if (expo_factor > q_coeff[0]) expo_factor = q_coeff[0];
            else if (expo_factor < q_coeff[n_coeff-1]) {
                if (zero_center) {
                    tran_weights[f*size + i] = 0;
                    continue;
                } else {
                    expo_factor = q_coeff[n_coeff-1];
                }
            }
            tran_weights[f*size + i] = -mean * powf(2.f, expo_factor);
        }
    }
}

void make_shifting_weights_mean_gpu(float *weights, int n, int size, float *tran_weights, float* q_coeff, int n_coeff, int zero_center)
{
    make_shifting_weights_mean_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, tran_weights, q_coeff, n_coeff, zero_center);
    check_error(hipPeekAtLastError());
}

__global__ void make_shifting_weights_kernel(float *weights, int n, float *tran_weights, float* q_coeff, int n_coeff, int zero_center)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;

    if (weights[i] > 0) {
        float expo_factor = roundf(log2f(weights[i]));
        if (expo_factor > q_coeff[0]) expo_factor = q_coeff[0];
        else if (expo_factor < q_coeff[n_coeff-1]) {
            if (zero_center) {
                tran_weights[i] = 0;
                return;
            } else {
                expo_factor = q_coeff[n_coeff-1];
            }
        }
        tran_weights[i] = powf(2.f, expo_factor);
    } else {
        float expo_factor = roundf(log2f(-weights[i]));
        if (expo_factor > q_coeff[0]) expo_factor = q_coeff[0];
        else if (expo_factor < q_coeff[n_coeff-1]) {
            if (zero_center) {
                tran_weights[i] = 0;
                return;
            } else {
                expo_factor = q_coeff[n_coeff-1];
            }
        }
        tran_weights[i] = -powf(2.f, expo_factor);
    }
}

void make_shifting_weights_gpu(float *weights, int n, float *tran_weights, float* q_coeff, int n_coeff, int zero_center)
{
    make_shifting_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, tran_weights, q_coeff, n_coeff, zero_center);
    check_error(hipPeekAtLastError());
}

__global__ void uniform_quantize_weights_kernel(float *weights, int n, float *tran_weights, float step_size, float* q_coeff, int n_coeff, int zero_center)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;

    tran_weights[i] = (round(weights[i]/step_size + 0.5 - (float)zero_center/2) - 0.5 + (float)zero_center/2) * step_size;
    tran_weights[i] = tran_weights[i] > q_coeff[n_coeff - 1] ? q_coeff[n_coeff - 1] : (tran_weights[i] < -q_coeff[n_coeff - 1] ? -q_coeff[n_coeff - 1] : tran_weights[i]);
}

void uniform_quantize_weights_gpu(float *weights, int n, float *tran_weights, float step_size, float* q_coeff, int n_coeff, int zero_center)
{
    uniform_quantize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, tran_weights, step_size, q_coeff, n_coeff, zero_center);
    check_error(hipPeekAtLastError());
}

__global__ void make_cycle_weights_kernel(float* weights, int n, float* tran_weights, int num_level, float step)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float pos_pole = powf(2.f, num_level - 1) - 1;
    float neg_pole = -powf(2.f, num_level - 1);

    float period = powf(2.f, num_level);
    tran_weights[i] = round(weights[i] / step);
    tran_weights[i] = tran_weights[i] - (int)(tran_weights[i] / period) * period;
    if (tran_weights[i] > pos_pole) {
        tran_weights[i] -= period;
    } else if (tran_weights[i] < neg_pole) {
        tran_weights[i] += period;
    }
    tran_weights[i] *= step;
}

void make_cycle_weights_gpu(float* weights, int n, float* tran_weights, int num_level, float step)
{
    make_cycle_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, tran_weights, num_level, step);
    check_error(hipPeekAtLastError());
}